#include "hip/hip_runtime.h"
#include <miner.h>

#include <cuda_helper.h>



#define saes_data(w) {\
    w(0x63), w(0x7c), w(0x77), w(0x7b), w(0xf2), w(0x6b), w(0x6f), w(0xc5),\
    w(0x30), w(0x01), w(0x67), w(0x2b), w(0xfe), w(0xd7), w(0xab), w(0x76),\
    w(0xca), w(0x82), w(0xc9), w(0x7d), w(0xfa), w(0x59), w(0x47), w(0xf0),\
    w(0xad), w(0xd4), w(0xa2), w(0xaf), w(0x9c), w(0xa4), w(0x72), w(0xc0),\
    w(0xb7), w(0xfd), w(0x93), w(0x26), w(0x36), w(0x3f), w(0xf7), w(0xcc),\
    w(0x34), w(0xa5), w(0xe5), w(0xf1), w(0x71), w(0xd8), w(0x31), w(0x15),\
    w(0x04), w(0xc7), w(0x23), w(0xc3), w(0x18), w(0x96), w(0x05), w(0x9a),\
    w(0x07), w(0x12), w(0x80), w(0xe2), w(0xeb), w(0x27), w(0xb2), w(0x75),\
    w(0x09), w(0x83), w(0x2c), w(0x1a), w(0x1b), w(0x6e), w(0x5a), w(0xa0),\
    w(0x52), w(0x3b), w(0xd6), w(0xb3), w(0x29), w(0xe3), w(0x2f), w(0x84),\
    w(0x53), w(0xd1), w(0x00), w(0xed), w(0x20), w(0xfc), w(0xb1), w(0x5b),\
    w(0x6a), w(0xcb), w(0xbe), w(0x39), w(0x4a), w(0x4c), w(0x58), w(0xcf),\
    w(0xd0), w(0xef), w(0xaa), w(0xfb), w(0x43), w(0x4d), w(0x33), w(0x85),\
    w(0x45), w(0xf9), w(0x02), w(0x7f), w(0x50), w(0x3c), w(0x9f), w(0xa8),\
    w(0x51), w(0xa3), w(0x40), w(0x8f), w(0x92), w(0x9d), w(0x38), w(0xf5),\
    w(0xbc), w(0xb6), w(0xda), w(0x21), w(0x10), w(0xff), w(0xf3), w(0xd2),\
    w(0xcd), w(0x0c), w(0x13), w(0xec), w(0x5f), w(0x97), w(0x44), w(0x17),\
    w(0xc4), w(0xa7), w(0x7e), w(0x3d), w(0x64), w(0x5d), w(0x19), w(0x73),\
    w(0x60), w(0x81), w(0x4f), w(0xdc), w(0x22), w(0x2a), w(0x90), w(0x88),\
    w(0x46), w(0xee), w(0xb8), w(0x14), w(0xde), w(0x5e), w(0x0b), w(0xdb),\
    w(0xe0), w(0x32), w(0x3a), w(0x0a), w(0x49), w(0x06), w(0x24), w(0x5c),\
    w(0xc2), w(0xd3), w(0xac), w(0x62), w(0x91), w(0x95), w(0xe4), w(0x79),\
    w(0xe7), w(0xc8), w(0x37), w(0x6d), w(0x8d), w(0xd5), w(0x4e), w(0xa9),\
    w(0x6c), w(0x56), w(0xf4), w(0xea), w(0x65), w(0x7a), w(0xae), w(0x08),\
    w(0xba), w(0x78), w(0x25), w(0x2e), w(0x1c), w(0xa6), w(0xb4), w(0xc6),\
    w(0xe8), w(0xdd), w(0x74), w(0x1f), w(0x4b), w(0xbd), w(0x8b), w(0x8a),\
    w(0x70), w(0x3e), w(0xb5), w(0x66), w(0x48), w(0x03), w(0xf6), w(0x0e),\
    w(0x61), w(0x35), w(0x57), w(0xb9), w(0x86), w(0xc1), w(0x1d), w(0x9e),\
    w(0xe1), w(0xf8), w(0x98), w(0x11), w(0x69), w(0xd9), w(0x8e), w(0x94),\
    w(0x9b), w(0x1e), w(0x87), w(0xe9), w(0xce), w(0x55), w(0x28), w(0xdf),\
    w(0x8c), w(0xa1), w(0x89), w(0x0d), w(0xbf), w(0xe6), w(0x42), w(0x68),\
    w(0x41), w(0x99), w(0x2d), w(0x0f), w(0xb0), w(0x54), w(0xbb), w(0x16) }

#define SAES_WPOLY           0x011b

#define saes_b2w(b0, b1, b2, b3) (((uint32_t)(b3) << 24) | \
    ((uint32_t)(b2) << 16) | ((uint32_t)(b1) << 8) | (b0))

#define saes_f2(x)   ((x<<1) ^ (((x>>7) & 1) * SAES_WPOLY))
#define saes_f3(x)   (saes_f2(x) ^ x)
#define saes_h0(x)   (x)

#define saes_u0(p)   saes_b2w(saes_f2(p),          p,          p, saes_f3(p))
#define saes_u1(p)   saes_b2w(saes_f3(p), saes_f2(p),          p,          p)
#define saes_u2(p)   saes_b2w(         p, saes_f3(p), saes_f2(p),          p)
#define saes_u3(p)   saes_b2w(         p,          p, saes_f3(p), saes_f2(p))

static __constant__  uint32_t saes_table[4][256] = { saes_data(saes_u0), saes_data(saes_u1), saes_data(saes_u2), saes_data(saes_u3) };



typedef uint4 uint128m;
#define GPU_DEBUG
#define VERUS_KEY_SIZE 8832
#define VERUS_KEY_SIZE128 552
#define THREADS 128


#define AES2_EMU(s0, s1, rci) \
  aesenc(&s0, &rc[rci],sharedMemory1); \
  aesenc(&s1, &rc[rci + 1],sharedMemory1); \
  aesenc(&s0, &rc[rci + 2],sharedMemory1); \
  aesenc(&s1, &rc[rci + 3],sharedMemory1);

#define AES4(s0, s1, s2, s3, rci) \
  aesenc(&s0, &rc[rci],sharedMemory1); \
  aesenc(&s1, &rc[rci + 1],sharedMemory1); \
  aesenc(&s2, &rc[rci + 2],sharedMemory1); \
  aesenc(&s3, &rc[rci + 3],sharedMemory1); \
  aesenc(&s0, &rc[rci + 4], sharedMemory1); \
  aesenc(&s1, &rc[rci + 5], sharedMemory1); \
  aesenc(&s2, &rc[rci + 6], sharedMemory1); \
  aesenc(&s3, &rc[rci + 7], sharedMemory1);


#define AES4_LAST(s3, rci) \
  aesenc(&s3, &rc[rci + 2],sharedMemory1); \
  aesenc(&s3, &rc[rci + 6], sharedMemory1); \


#define TRUNCSTORE(out, s4) \
  *(uint32_t*)(out + 28) = s4.y;

#define MIX2_EMU(s0, s1) \
  tmp = _mm_unpacklo_epi32_emu(s0, s1); \
  s1 = _mm_unpackhi_epi32_emu(s0, s1); \
  s0 = tmp;

#define MIX4(s0, s1, s2, s3) \
  tmp  = _mm_unpacklo_epi32_emu(s0, s1); \
  s0 = _mm_unpackhi_epi32_emu(s0, s1); \
  s1 = _mm_unpacklo_epi32_emu(s2, s3); \
  s2 = _mm_unpackhi_epi32_emu(s2, s3); \
  s3 = _mm_unpacklo_epi32_emu(s0, s2); \
  s0 = _mm_unpackhi_epi32_emu(s0, s2); \
  s2 = _mm_unpackhi_epi32_emu(s1, tmp); \
  s1 = _mm_unpacklo_epi32_emu(s1, tmp);

#define MIX4_LASTBUT1(s0, s1, s2, s3) \
  tmp  = _mm_unpacklo_epi32_emu(s0, s1); \
  s1 = _mm_unpacklo_epi32_emu(s2, s3); \
  s2 = _mm_unpackhi_epi32_emu(s1, tmp); 


__host__ void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id, uint32_t throughput);
__global__ void verus_gpu_hash(const uint32_t threads, const uint32_t startNonce, uint32_t * __restrict__ resNonce,
	uint128m * __restrict__ d_key_input, uint8_t version);
__global__ void verus_extra_gpu_prepare(const uint32_t threads, uint128m * d_key_input);

#define TOTAL_MAX 0x10000

static uint32_t *d_nonces[MAX_GPUS];
static uint4 *d_long_keys[MAX_GPUS];



__device__ __constant__ uint128m vkey[VERUS_KEY_SIZE128];
__device__ __constant__ uint128m blockhash_half[4];
__device__ __constant__ uint32_t ptarget[8];

__host__
void verus_init(int thr_id, uint32_t throughput)
{
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(verus_gpu_hash), hipFuncCachePreferEqual);
	CUDA_SAFE_CALL(hipMalloc(&d_nonces[thr_id], 1 * sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMalloc(&d_long_keys[thr_id], TOTAL_MAX * VERUS_KEY_SIZE));

};

__host__
void verus_setBlock(uint8_t *blockf, uint32_t *pTargetIn, uint8_t *lkey, int thr_id, uint32_t throughput)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ptarget), (void**)pTargetIn, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(blockhash_half), (void**)blockf, 64 * sizeof(uint8_t), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(vkey), (void**)lkey, VERUS_KEY_SIZE * sizeof(uint8_t), 0, hipMemcpyHostToDevice));



};
__host__
void verus_hash(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *resNonces, uint8_t version)
{
	hipMemset(d_nonces[thr_id], 0xff, 1 * sizeof(uint32_t));
	const uint32_t threadsperblock = THREADS;
	//	const uint32_t threadsperblock2 = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 grid2(threads);
	dim3 block(threadsperblock);

	//verus_extra_gpu_prepare << <grid2, 128,0, streams[thr_id][0] >> > (0, d_long_keys[thr_id]); //setup global mem with lots of keys	
	verus_gpu_hash << <grid, block >> >(threads, startNonce, d_nonces[thr_id], d_long_keys[thr_id], version);
	CUDA_SAFE_CALL(hipMemcpy(resNonces, d_nonces[thr_id], 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));

};

#define _mm_xor_si128_emu(a,b) a^b;

__device__   uint128m _mm_clmulepi64_si128_emu(uint128m ai, uint128m bi, int imm)
{
	uint64_t a = ((uint64_t*)&ai)[0]; 

	uint64_t b = ((uint64_t*)&bi)[1];

	uint64_t r[2]; 

	if (__popcll(a) > __popcll(b)) 
	{
		a = b; b = ((uint64_t*)&ai)[0];
	}
	r[0] = 0; r[1] = 0;

	uint64_t w = a; int counter = 0; int first;

	while ((first = __clzll(w) + 1) != 65) {
		w <<= (first);
		counter += (first);

		r[0] ^= b << (64 - counter);
		r[1] ^= b >> ((counter));
	};

	return ((uint128m*)&r)[0];
}

__device__  __forceinline__ uint128m _mm_clmulepi64_si128_emu2(uint128m ai)
{
	uint64_t a = ((uint64_t*)&ai)[1];

	uint64_t result[2] = { 0,0 };
	result[0] = a;
	result[0] ^= a << 1;
	result[1] ^= a >> 63;
	result[0] ^= a << 3;
	result[1] ^= a >> 61;
	result[0] ^= a << 4;
	result[1] ^= a >> 60;

	return AS_UINT4(result);
}

#define _mm_load_si128_emu(p) (*(uint128m*)(p));

#define _mm_cvtsi128_si64_emu(p) (((int64_t *)&p)[0]);

#define _mm_cvtsi128_si32_emu(p) (((int32_t *)&a)[0]);


__device__   __forceinline__   void _mm_unpackboth_epi32_emu(uint128m &a, uint128m &b)
{
	uint64_t value;

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(a.z), "r"(a.y));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.y), "=r"(a.z) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.x), "r"(a.y));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.y), "=r"(b.x) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.z), "r"(a.w));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.w), "=r"(b.z) : "l"(value));

	asm("mov.b64 %0, {%1, %2}; ": "=l"(value) : "r"(b.y), "r"(a.w));
	asm("mov.b64 {%0, %1}, %2; ": "=r"(a.w), "=r"(b.y) : "l"(value));
}

__device__  __forceinline__ uint128m unpackandmix(uint128m a, uint128m b, uint128m acc)
{
	uint128m tmp;

	tmp.x = a.x ^ acc.x ^ a.z;
	tmp.y = b.x ^ acc.y ^ b.z;
	tmp.z = a.y ^ acc.z ^ a.w;
	tmp.w = b.y ^ acc.w ^ b.w;

	return tmp;
}

__device__  __forceinline__ uint128m _mm_unpacklo_epi32_emu(uint128m a, uint128m b)
{

	//uint4 t;

	//	t.x = a.x;
	a.z = a.y;
	a.y = b.x;
	a.w = b.y;
	return a;
}

__device__  __forceinline__ uint128m _mm_unpackhi_epi32_emu(uint128m a, uint128m b)
{

	//uint4 t;
	b.x = a.z;
	b.y = b.z;
	b.z = a.w;
	//t.w = b.w;

	return b;
}


__device__   void aesenc(uint4 * __restrict__ ptr, const uint128m * __restrict__ key, uint32_t * __restrict__ t)
{
	//#define XT(x) (((x) << 1) ^ (((x) >> 7) ? 0x1b : 0))

	//#define XT4(x) ((((x) << 1) & 0xfefefefe) ^ ((((x) >> 31) & 1) ? 0x1b000000 : 0)^ ((((x) >> 23)&1) ? 0x001b0000 : 0)^ ((((x) >> 15)&1) ? 0x00001b00 : 0)^ ((((x) >> 7)&1) ? 0x0000001b : 0))
	uint32_t x0 = ptr[0].x;
	uint32_t x1 = ptr[0].y;
	uint32_t x2 = ptr[0].z;
	uint32_t x3 = ptr[0].w;

	uint32_t y0 = t[x0 & 0xff]; x0 >>= 8;
	uint32_t y1 = t[x1 & 0xff]; x1 >>= 8;
	uint32_t y2 = t[x2 & 0xff]; x2 >>= 8;
	uint32_t y3 = t[x3 & 0xff]; x3 >>= 8;
	t += 256;

	y0 ^= t[x1 & 0xff]; x1 >>= 8;
	y1 ^= t[x2 & 0xff]; x2 >>= 8;
	y2 ^= t[x3 & 0xff]; x3 >>= 8;
	y3 ^= t[x0 & 0xff]; x0 >>= 8;
	t += 256;

	y0 ^= t[x2 & 0xff]; x2 >>= 8;
	y1 ^= t[x3 & 0xff]; x3 >>= 8;
	y2 ^= t[x0 & 0xff]; x0 >>= 8;
	y3 ^= t[x1 & 0xff]; x1 >>= 8;
	t += 256;

	y0 ^= t[x3];
	y1 ^= t[x0];
	y2 ^= t[x1];
	y3 ^= t[x2];

	ptr[0].x = y0 ^ key[0].x;
	ptr[0].y = y1 ^ key[0].y;
	ptr[0].z = y2 ^ key[0].z;
	ptr[0].w = y3 ^ key[0].w;

}


__device__  __forceinline__ uint128m _mm_cvtsi32_si128_emu(uint32_t lo)
{
	uint128m result = { 0 };
	result.x = lo;

	return result;
}
__device__  __forceinline__ uint128m _mm_cvtsi64_si128_emu(uint64_t lo)
{
	uint128m result = { 0 };
	((uint64_t *)&result)[0] = lo;
	//((uint64_t *)&result)[1] = 0;
	return result;
}
__device__  __forceinline__ uint128m _mm_set_epi64x_emu(uint64_t hi, uint64_t lo)
{
	uint128m result;
	((uint64_t *)&result)[0] = lo;
	((uint64_t *)&result)[1] = hi;
	return result;
}
__device__ __forceinline__ uint128m _mm_shuffle_epi8_emu(uint2 b)
{
	uint128m result = { 0 };
	const uint128m M = { 0x2d361b00,0x415a776c,0xf5eec3d8,0x9982afb4 };
	const uint2 Q = { 0x80808080, 0x80808080};
	const uint2 W = b & Q;


#pragma unroll
	for (int i = 0; i < 8; i++)
	{
		if (!((uint8_t *)&W)[i])
		{
			((uint8_t *)&result)[i] = ((uint8_t *)&M)[((uint8_t *)&b)[i] & 0xf];
		}
	}

	return result;
}



__device__  __forceinline__ uint2 _mm_srli_si128_emu(uint128m input, int imm8)
{
	//we can cheat here as its an 8 byte shift just copy the 64bits
	uint2 temp;
	((uint64_t*)&temp)[0] = ((uint64_t*)&input)[1];
//	((uint64_t*)&temp)[1] = 0;


	return temp;
}



__device__    __forceinline__  uint128m _mm_mulhrs_epi16_emu(uint128m _a, uint128m _b)
{
	int16_t result[8];

	int32_t po;
	int16_t *a = (int16_t*)&_a, *b = (int16_t*)&_b;
#pragma nounroll
	for (int i = 0; i < 8; i++)
	{
		asm("mad.lo.s32 %0, %1, %2, 16384; ": "=r"(po) : "r"((int32_t)a[i]), "r"((int32_t)b[i]));
		result[i] = po >> 15;
	}

	return *(uint128m *)result;
}


__device__    __forceinline__  void case_0(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prandex;

	const uint128m temp2 = _mm_load_si128_emu(pbuf - ((selector & 1) ? 1 : -1));


	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prand;
	prand = tempa2;


	const uint128m temp22 = _mm_load_si128_emu(pbuf);
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
	acc = _mm_xor_si128_emu(clprod12, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prandex = tempb2;

}

__device__   __forceinline__  void case_4(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prand;
	const uint128m temp2 = _mm_load_si128_emu(pbuf);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);
	const uint128m clprod2 = _mm_clmulepi64_si128_emu(temp2, temp2, 0x10);
	acc = _mm_xor_si128_emu(clprod2, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prandex;
	prandex = tempa2;

	const uint128m temp22 = _mm_load_si128_emu(pbuf - ((selector & 1) ? 1 : -1));
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	acc = _mm_xor_si128_emu(add12, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prand = tempb2;
}

__device__    __forceinline__  void case_8(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prandex;
	const uint128m temp2 = _mm_load_si128_emu(pbuf);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	acc = _mm_xor_si128_emu(add1, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);

	const uint128m temp12 = prand;
	prand = tempa2;

	const uint128m temp22 = _mm_load_si128_emu(pbuf - ((selector & 1) ? 1 : -1));
	const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
	const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
	acc = _mm_xor_si128_emu(clprod12, acc);
	const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
	acc = _mm_xor_si128_emu(clprod22, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
	prandex = tempb2;
}



__device__   __forceinline__  void case_0c_1(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prand;
	const uint128m temp2 = _mm_load_si128_emu(pbuf - ((selector & 1) ? 1 : -1));
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

	// cannot be zero here
	const int32_t divisor = ((uint32_t*)&selector)[0];

	acc = _mm_xor_si128_emu(add1, acc);

	int64_t dividend = _mm_cvtsi128_si64_emu(acc);
	int64_t tmpmod = dividend % divisor;
	const uint128m modulo = _mm_cvtsi32_si128_emu(tmpmod);
	acc = _mm_xor_si128_emu(modulo, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);
	dividend &= 1;
	if (dividend)
	{
		const uint128m temp12 = prandex;
		prandex = tempa2;

		const uint128m temp22 = _mm_load_si128_emu(pbuf);
		const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
		const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
		acc = _mm_xor_si128_emu(clprod12, acc);
		const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
		acc = _mm_xor_si128_emu(clprod22, acc);

		const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
		const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
		prand = tempb2;
	}
	else
	{
		const uint128m tempb3 = prandex;
		prandex = tempa2;
		prand = tempb3;
	}
}

__device__   __forceinline__  void case_0c_2(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = prand;
	const uint128m temp2 = _mm_load_si128_emu(pbuf - ((selector & 1) ? 1 : -1));
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);

	// cannot be zero here
	const int32_t divisor = ((uint32_t*)&selector)[0];

	acc = _mm_xor_si128_emu(add1, acc);

	int64_t dividend = _mm_cvtsi128_si64_emu(acc);
	int64_t tmpmod = dividend % divisor;
	const uint128m modulo = _mm_cvtsi32_si128_emu(tmpmod);
	acc = _mm_xor_si128_emu(modulo, acc);

	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp1);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp1);
	dividend &= 1;
	if (dividend)
	{
		const uint128m temp12 = prandex;
		prandex = tempa2;

		const uint128m temp22 = _mm_load_si128_emu(pbuf);
		const uint128m add12 = _mm_xor_si128_emu(temp12, temp22);
		const uint128m clprod12 = _mm_clmulepi64_si128_emu(add12, add12, 0x10);
		acc = _mm_xor_si128_emu(clprod12, acc);
		const uint128m clprod22 = _mm_clmulepi64_si128_emu(temp22, temp22, 0x10);
		acc = _mm_xor_si128_emu(clprod22, acc);

		const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, temp12);
		const uint128m tempb2 = _mm_xor_si128_emu(tempb1, temp12);
		prand = tempb2;
	}
	else
	{
		const uint128m tempb3 = prandex;
		prandex = tempa2;
		prand = tempb3;
		const uint128m tempb4 = _mm_load_si128_emu(pbuf);
		acc = _mm_xor_si128_emu(tempb4, acc);
	}
}

__device__   __forceinline__  void case_10(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc, uint128m *rc, uint32_t prand_idx, uint32_t *sharedMemory1)
{			// a few AES operations

	uint128m tmp;

	uint128m temp1 = _mm_load_si128_emu(pbuf - ((selector & 1) ? 1 : -1));
	uint128m temp2 = _mm_load_si128_emu(pbuf);

	AES2_EMU(temp1, temp2, 0);
	MIX2_EMU(temp1, temp2);


	AES2_EMU(temp1, temp2, 4);
	MIX2_EMU(temp1, temp2);

	AES2_EMU(temp1, temp2, 8);
	acc = unpackandmix(temp1, temp2, acc);

	const uint128m tempa1 = prand;
	const uint128m tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
	const uint128m tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

	const uint128m tempa4 = prandex;
	prandex = tempa3;
	prand = tempa4;
}
__device__   __forceinline__  void case_14(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc, uint128m *randomsource, uint32_t prand_idx, uint32_t *sharedMemory1)
{
	// we'll just call this one the monkins loop, inspired by Chris
	const uint128m *buftmp = pbuf - ((selector & 1) ? 1 : -1);
	uint128m tmp; // used by MIX2

	uint64_t rounds = selector >> 61; // loop randomly between 1 and 8 times
	uint128m *rc = &randomsource[prand_idx];

	uint64_t aesround = 0;
	uint128m onekey;
	uint64_t loop_c;

	do {
		loop_c = selector & ((uint64_t)0x10000000 << rounds);
		if (loop_c)
		{
			onekey = _mm_load_si128_emu(rc++);
			const uint128m temp2 = _mm_load_si128_emu(rounds & 1 ? pbuf : buftmp);
			const uint128m add1 = _mm_xor_si128_emu(onekey, temp2);
			const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
			acc = _mm_xor_si128_emu(clprod1, acc);
			rounds--;
			if (rounds != (uint64_t)0xffffffffffffffff)	loop_c = selector & ((uint64_t)0x10000000 << rounds);
		}
		if (!loop_c && (rounds != (uint64_t)0xffffffffffffffff))
		{
			onekey = _mm_load_si128_emu(rc++);
			uint128m temp2 = _mm_load_si128_emu(rounds & 1 ? buftmp : pbuf);

			const uint64_t roundidx = aesround++ << 2;
			AES2_EMU(onekey, temp2, roundidx);
			acc = unpackandmix(onekey, temp2, acc);

			rounds--;
		}
	} while (rounds != (uint64_t)0xffffffffffffffff);

	const uint128m tempa1 = (prand);
	const uint128m tempa2 = _mm_mulhrs_epi16_emu(acc, tempa1);
	const uint128m tempa3 = _mm_xor_si128_emu(tempa1, tempa2);

	const uint128m tempa4 = (prandex);
	prandex = tempa3;
	prand = tempa4;
}

__device__   __forceinline__  void  case_18_1(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc, uint128m *randomsource, uint32_t prand_idx)
{
	// we'll just call this one the monkins loop, inspired by Chris
	const uint4 *buftmp = pbuf - ((selector & 1) ? 1 : -1);


	uint64_t rounds = selector >> 61; // loop randomly between 1 and 8 times
	uint4 *rc = &randomsource[prand_idx];

	uint4 onekey;
	uint64_t loop_c;

	do {
		loop_c = selector & ((uint64_t)0x10000000 << rounds);
		if (loop_c)
		{
			onekey = _mm_load_si128_emu(rc++);
			const uint4 temp2 = _mm_load_si128_emu(rounds & 1 ? pbuf : buftmp);
			const uint4 add1 = _mm_xor_si128_emu(onekey, temp2);

			const int32_t divisor = (uint32_t)selector;
			const int64_t dividend = ((int64_t*)&add1)[0];
			uint4 modulo = { 0 }; ((int32_t*)&modulo)[0] = (dividend % divisor);
			acc = modulo ^ acc;
			rounds--;
			if (rounds != (uint64_t)0xffffffffffffffff)	loop_c = selector & ((uint64_t)0x10000000 << rounds);
		}
		if (!loop_c && (rounds != (uint64_t)0xffffffffffffffff))
		{
			onekey = _mm_load_si128_emu(rc++);
			uint4 temp2 = _mm_load_si128_emu(rounds & 1 ? buftmp : pbuf);
			uint4 add1 = (onekey^ temp2);
			uint4 clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0);
			uint4 clprod2 = _mm_mulhrs_epi16_emu(acc, clprod1);
			acc = clprod2^ acc;
			rounds--;
		}
	} while (rounds != (uint64_t)0xffffffffffffffff);

	const uint4 tempa3 = (prandex);
	const uint4 tempa4 = _mm_xor_si128_emu(tempa3, acc);
	prandex = tempa4;
	prand = onekey;
}

__device__   __forceinline__  void  case_18_2(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc, uint128m *randomsource, uint32_t prand_idx)
{
	// we'll just call this one the monkins loop, inspired by Chris
	const uint4 *buftmp = pbuf - ((selector & 1) ? 1 : -1);


	uint64_t rounds = selector >> 61; // loop randomly between 1 and 8 times
	uint4 *rc = &randomsource[prand_idx];

	uint4 onekey;
	uint64_t loop_c;

	do {
		loop_c = selector & ((uint64_t)0x10000000 << rounds);
		if (loop_c)
		{
			onekey = _mm_load_si128_emu(rc++);
			const uint4 temp2 = _mm_load_si128_emu(rounds & 1 ? pbuf : buftmp);
			onekey = _mm_xor_si128_emu(onekey, temp2);

			const int32_t divisor = (uint32_t)selector;
			const int64_t dividend = ((int64_t*)&onekey)[0];
			uint4 modulo = { 0 }; ((int32_t*)&modulo)[0] = (dividend % divisor);
			acc = modulo ^ acc;
			rounds--;
			if (rounds != (uint64_t)0xffffffffffffffff)	loop_c = selector & ((uint64_t)0x10000000 << rounds);
		}
		if (!loop_c && (rounds != (uint64_t)0xffffffffffffffff))
		{
			onekey = _mm_load_si128_emu(rc++);
			uint4 temp2 = _mm_load_si128_emu(rounds & 1 ? buftmp : pbuf);
			uint4 add1 = (onekey^ temp2);
			onekey = _mm_clmulepi64_si128_emu(add1, add1, 0);
			uint4 clprod2 = _mm_mulhrs_epi16_emu(acc, onekey);
			acc = clprod2^ acc;
			rounds--;
		}
	} while (rounds != (uint64_t)0xffffffffffffffff);

	const uint4 tempa3 = (prandex);
	const uint4 tempa4 = _mm_xor_si128_emu(tempa3, acc);
	prandex = onekey;
	prand = tempa4;
}

__device__    __forceinline__   void case_1c_1(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = _mm_load_si128_emu(pbuf);
	const uint128m temp2 = (prandex);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);


	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp2);
	const uint128m tempa3 = (prand);


	prand = tempa2;

	acc = _mm_xor_si128_emu(tempa3, acc);

	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, tempa3);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, tempa3);
	prandex = tempb2;
}

__device__    __forceinline__   void case_1c_2(uint128m &prand, uint128m &prandex, const  uint128m *pbuf,
	uint64_t selector, uint128m &acc)
{
	const uint128m temp1 = _mm_load_si128_emu(pbuf);
	const uint128m temp2 = (prandex);
	const uint128m add1 = _mm_xor_si128_emu(temp1, temp2);
	const uint128m clprod1 = _mm_clmulepi64_si128_emu(add1, add1, 0x10);
	acc = _mm_xor_si128_emu(clprod1, acc);


	const uint128m tempa1 = _mm_mulhrs_epi16_emu(acc, temp2);
	const uint128m tempa2 = _mm_xor_si128_emu(tempa1, temp2);
	const uint128m tempa3 = (prand);


	prand = tempa2;

	acc = _mm_xor_si128_emu(tempa3, acc);
	const uint128m temp4 = _mm_load_si128_emu(pbuf - ((selector & 1) ? 1 : -1));
	acc = _mm_xor_si128_emu(temp4, acc);
	const uint128m tempb1 = _mm_mulhrs_epi16_emu(acc, tempa3);
	const uint128m tempb2 = _mm_xor_si128_emu(tempb1, tempa3);
	prandex = tempb2;
}

__device__   __forceinline__ uint2 precompReduction64(uint128m A) {


	//static const uint128m M = { 0x2d361b00,0x415a776c,0xf5eec3d8,0x9982afb4 };
	// const uint128m tmp = { 27 };
	// A.z = 0;
	//tmp.x = 27u;
	uint128m Q2 = _mm_clmulepi64_si128_emu2(A);
	uint128m Q3 = _mm_shuffle_epi8_emu({ Q2.z,Q2.w });

	//uint128m Q4 = _mm_xor_si128_emu(Q2, A);
	uint2 final;
	final.x = xor3(A.x, Q2.x, Q3.x);
	final.y = xor3(A.y, Q2.y, Q3.y);

	return final;
}


#define PRE			selector = _mm_cvtsi128_si64_emu(acc);\
			if (i > 0) {\
				prand_idx = ((acc.x >> 5) & 511);\
				prandex_idx = ((acc.y) & 511);\
				prand = randomsource[prand_idx];\
				prandex = randomsource[prandex_idx];\
			}\
			pbuf = buf + (acc.x & 3);\
			case_v = selector & 0x1cu;

#define PRE2			selector = _mm_cvtsi128_si64_emu(acc);\
			if (i > 0) {\
				prand_idx = ((acc.x >> 5) & 511);\
				prandex_idx = ((acc.y) & 511);\
				prand = randomsource[prand_idx];\
				prandex = randomsource[prandex_idx];\
			}\
			pbuf = buf + (acc.x & 3);\
			case_v = selector & 0x1cu;

__device__   __forceinline__  uint2 __verusclmulwithoutreduction64alignedrepeatgpu(uint128m * __restrict__ randomsource, const  uint128m *  __restrict__  buf,
	uint32_t *  __restrict__ sharedMemory1, uint8_t version)
{
	uint128m const *pbuf;
	//keyMask >>= 4;
	uint128m acc = vkey[513];


	// divide key mask by 32 from bytes to uint128m

	uint16_t prand_idx, prandex_idx;
	uint64_t selector;
	uint128m prand;
	uint128m prandex;
	prand_idx = ((acc.x >> 5) & 511);
	prandex_idx = ((acc.y) & 511);

	prand = vkey[prand_idx];
	prandex = vkey[prandex_idx];
	//#pragma unroll
	int i = 0;
	uint8_t case_v;
	selector = _mm_cvtsi128_si64_emu(acc);
	pbuf = buf + (acc.x & 3);
		case_v = selector & 0x1cu;
		do
		{



			if (((case_v == 0x14)))
			{
				case_14(prand, prandex, pbuf, selector, acc, randomsource, prand_idx, sharedMemory1);

				randomsource[prand_idx] = prand;
				randomsource[prandex_idx] = prandex;
				i++;
				if (i == 32)break;
				PRE

			}

			if ((case_v == 0x10))
			{
				uint128m *rc = &randomsource[prand_idx];
				case_10(prand, prandex, pbuf, selector, acc, rc, prand_idx, sharedMemory1);

				randomsource[prand_idx] = prand;
				randomsource[prandex_idx] = prandex;
				i++;
				if (i == 32)break;
				PRE
			}
			if (case_v == 0)
			{

				case_0(prand, prandex, pbuf, selector, acc);

				randomsource[prand_idx] = prand;
				randomsource[prandex_idx] = prandex;
				i++;
				if (i == 32)break;
				PRE

			}
			if (case_v == 4)
			{
				case_4(prand, prandex, pbuf, selector, acc);

				randomsource[prand_idx] = prand;
				randomsource[prandex_idx] = prandex;
				i++;
				if (i == 32)break;
				PRE
			}
			if (case_v == 8)
			{
				case_8(prand, prandex, pbuf, selector, acc);

				randomsource[prand_idx] = prand;
				randomsource[prandex_idx] = prandex;
				i++;
				if (i == 32)break;
				PRE

			}
			if (case_v == 0xc)
			{
				if (version == 3)
					case_0c_1(prand, prandex, pbuf, selector, acc);
				else
					case_0c_2(prand, prandex, pbuf, selector, acc);

				randomsource[prand_idx] = prand;
				randomsource[prandex_idx] = prandex;
				i++;
				if (i == 32)break;
				PRE

			}

			if (case_v == 0x18)
			{
				if (version == 3)
					case_18_1(prand, prandex, pbuf, selector, acc, randomsource, prand_idx);
				else
					case_18_2(prand, prandex, pbuf, selector, acc, randomsource, prand_idx);

				randomsource[prand_idx] = prand;
				randomsource[prandex_idx] = prandex;
				i++;
				if (i == 32)break;
				PRE

			}
			if (case_v == 0x1c)
			{
				if (version == 3)
					case_1c_1(prand, prandex, pbuf, selector, acc);
				else
					case_1c_2(prand, prandex, pbuf, selector, acc);

				randomsource[prand_idx] = prand;
				randomsource[prandex_idx] = prandex;
				i++;
				if (i == 32)break;
				PRE

			}



		} while (i != 32);
		acc.x ^= 0x00010000;

		return precompReduction64(acc);
}


__device__   __forceinline__  uint32_t haraka512_port_keyed2222(uint128m * __restrict__  in, uint128m * __restrict__  rc, uint32_t * __restrict__  sharedMemory1)
{
	uint128m s1, s2, s3, s4, tmp;

	s1 = in[0];
	s2 = in[1];
	s3 = in[2];
	s4 = in[3];

	AES4(s1, s2, s3, s4, 0);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 8);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 16);
	MIX4(s1, s2, s3, s4);

	AES4(s1, s2, s3, s4, 24);
	MIX4_LASTBUT1(s1, s2, s3, s4);


	AES4_LAST(s3, 32);

	return s3.z ^ in[3].y;

}



__global__ __launch_bounds__(THREADS, 1)
void verus_gpu_hash(const uint32_t threads, const uint32_t startNonce, uint32_t * __restrict__ resNonce,
	uint128m * __restrict__ d_key_input, uint8_t version)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	
	__shared__  uint128m j[4 * THREADS];
	uint128m *s = &j[threadIdx.x << 2];

	const uint32_t nounce = startNonce + thread;

	__shared__ uint32_t sharedMemory1[4][256];
	__shared__ uint128m sharedMemory3[VERUS_KEY_SIZE128];

	s[0] = blockhash_half[0];
	s[1] = blockhash_half[1];
	s[2] = blockhash_half[2];
	s[3] = blockhash_half[3];

	for (int i = threadIdx.x; i < 256; i += blockDim.x) {

		sharedMemory1[0][i] = saes_table[0][i];
		sharedMemory1[1][i] = saes_table[1][i];
		sharedMemory1[2][i] = saes_table[2][i];
		sharedMemory1[3][i] = saes_table[3][i];
	}

	for (int i = threadIdx.x; i < VERUS_KEY_SIZE128; i += blockDim.x) {

		sharedMemory3[i] = vkey[i];
	}
	__syncthreads();

	for (int i = 0; i < 512; i++) {

		d_key_input[(VERUS_KEY_SIZE128 * (thread & (TOTAL_MAX - 1))) + ((threadIdx.x + i) & 511)] = sharedMemory3[((threadIdx.x + i) & 511)];
	}
	int b = threadIdx.x % 40;
	for (int i = 0; i < 40; i++) {

		d_key_input[((VERUS_KEY_SIZE128 * (thread & (TOTAL_MAX - 1)))) + 512 + ((b) % 40)] = sharedMemory3[512 + ((b) % 40)]; b++;
	}
	s[2].x = nounce;
	s[0] = s[0] ^ s[2];
	s[1] = s[1] ^ s[3];


	uint2 acc = __verusclmulwithoutreduction64alignedrepeatgpu(&d_key_input[(VERUS_KEY_SIZE128 * (thread & (TOTAL_MAX - 1)))], s, sharedMemory1[0], version);

	s[0] = blockhash_half[0];
	s[1] = blockhash_half[1];

	uint2 tmp = ROR2(acc,8);
	s[3].x = tmp.x;
	s[3].y = tmp.y;
	s[3].z = tmp.x;
	s[3].w = tmp.y;

	s[2].w = (s[2].w & 0x00ffffff) | (acc.x & 0xff) << 24;
	acc.x &= 511;

	uint32_t hash = haraka512_port_keyed2222(s, (&d_key_input[(VERUS_KEY_SIZE128 * (thread & (TOTAL_MAX - 1)))] + acc.x), sharedMemory1[0]);
	if (hash < ptarget[7]) 
	{
		resNonce[0] = nounce;
	}


};

